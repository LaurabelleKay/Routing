#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <cctype>

#include "router.h"
#include "hip/hip_runtime.h"
#include ""

#include "common.h"

#ifdef DISPLAY
#include "display.h"
#endif

#define MAX_SHM 1024
#define EMPTY -2
#define TRUE 1
#define FALSE 0
#define EXPANDED -1
#define MAX_ATTEMPTS 10

//#define index(k, i, j) ((k) * (i) + (j))
#define min(a, b) (((a) < (b)) ? (a) : (b))
#define max(a, b) (((a) > (b)) ? (a) : (b))


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

bool toDelete(int n)
{
   return (n >= 4096);
}

__global__
void leeMoore(
   int srcx, 
   int srcy, 
   int snkx,
   int snky,
   int rTop,
   int rBottom, 
   int rLeft,
   int rRight,
   int gridx, 
   int gridy,
   int wire,
   int edgeIndex,
   int *success,
   int *graph
   //int *route
)
{
   int bid = blockIdx.x;
   int x = threadIdx.x;
   int y = threadIdx.y;

   __shared__ int frontier[MAX_SHM];
   __shared__ int costs[MAX_SHM];
   //__shared__ int tempCosts[MAX_SHM];
   __shared__ int from1[MAX_SHM];
   __shared__ int from2[MAX_SHM];
   __shared__ int done;

   //int rTop, int rBottom, int rLeft, int rRight; //Region boundaries

   int cost;

   int xg, yg; //Locations on the graph for this thread to read data from 
   xg = rLeft + x;
   yg = rTop + y;

   //printf("[%d][%d] = %d\n", xg, yg, graph[gridy * xg + yg]);

   int dimx = rRight - rLeft;
   int dimy = rBottom - rTop;

   //printf("[%d][%d] maps to [%d][%d]\n", x, y, xg, yg);

   done = 0; //FIXME: Don't think this will work!
   int count = 0;
   
   costs[dimy * x + y] = 1000;
   frontier[dimy * x + y] = FALSE;

   //Set the source as the frontier, and its cost as 0
   if(xg == srcx && yg == srcy)
   {
      frontier[dimy * x + y] = TRUE;
      costs[dimy * x + y] = 0;
   }

   from1[dimy * (x + 1) + y] = -30;
   from2[dimy * (x + 1) + y] = -30;
   if(x == 0 & y == 0)
   {
      success[edgeIndex] = 0;
   }

   __shared__ int foundSink;
   if(x == 0 && y == 0)
   {
      foundSink = 0;
   }
   
   //Source to sink propagation
   while(count++ < (dimx * dimy))
   {
      __syncthreads();
      if(foundSink == 1)
      {
         break;
      }
      
      if(frontier[dimy * x + y] == TRUE)
      {
         frontier[dimy * x + y] = EXPANDED;
         //printf("[%d][%d]([%d][%d]) frontier expanded. Cost: %d\n", x, y, xg, yg, costs[dimy * x + y]);
         //printf("[%d][%d] from [%d][%d]\n", x, y, from1[dimy * x + y], from2[dimy * x + y]);
         //atomicAdd(&done, 1);

         if(xg == snkx && yg == snky)
         {
            printf("Sink found! Cost is: %d\n", costs[dimy * x + y]);
            foundSink = 1;
            done = 1;
            atomicAdd(&done, 1E06);
            //printf("Done is now: %d\n", done);
         }

         //Assess top neighbour
         if(y > 0)
         {
            if((graph[gridy * xg + (yg - 1)] == -2 || graph[gridy * xg + (yg - 1)] == wire) && frontier[dimy * x  + (y - 1)] != EXPANDED) //Check for an obstruction
            {
               cost = costs[dimy * x + y] + 1;
               costs[dimy * x  + (y - 1)] = cost;
               frontier[dimy * x  + (y - 1)] = TRUE;
               from1[dimy * x  + (y - 1)] = x;
               from2[dimy * x  + (y - 1)] = y;
               //printf("[%d][%d]([%d][%d]) expands [%d][%d]([%d][%d])\n", x, y, xg, yg, x, y - 1, xg, yg - 1);
            }
         }

         //Assess bottom neighbour
         if(y < dimy - 1)
         {
            if((graph[gridy * xg + (yg + 1)] == -2 || graph[gridy * xg + (yg + 1)] == wire) && frontier[dimy * x  + (y  + 1)] != EXPANDED)
            {
               cost = costs[dimy * x + y] + 1;
               costs[dimy * x  + (y  + 1)] = cost;
               frontier[dimy * x  + (y  + 1)] = TRUE;
               from1[dimy * x  + (y  + 1)] = x;
               from2[dimy * x  + (y  + 1)] = y;
               //printf("[%d][%d]([%d][%d]) expands [%d][%d]([%d][%d])\n", x, y, xg, yg, x, y + 1, xg, yg + 1);
            }
         }

         //Assess left neighbour
         if(x < dimx - 1)
         {
            if((graph[gridy * (xg + 1) + yg] == -2 || graph[gridy * (xg + 1) + yg] == wire) && frontier[dimy * (x + 1) + y] != EXPANDED)
            {
               cost = costs[dimy * x + y] + 1;
               costs[dimy * (x + 1) + y] = cost;
               frontier[dimy * (x + 1) + y] = TRUE;
               from1[dimy * (x + 1) + y] = x;
               from2[dimy * (x + 1) + y] = y;
               //printf("[%d][%d]([%d][%d]) expands [%d][%d]([%d][%d])\n", x, y, xg, yg, x + 1, y, xg + 1, yg);
            }  
         }

         //Assess right neighbour
         if(x > 0)
         {
            if((graph[gridy * (xg - 1) + yg] == -2 || graph[gridy * (xg - 1) + yg] == wire) && frontier[dimy * (x - 1) + y] != EXPANDED)
            {
               cost = costs[dimy * x + y] + 1;
               costs[dimy * (x - 1) + y] = cost;
               frontier[dimy * (x - 1) + y] = TRUE;
               from1[dimy * (x - 1) + y] = x;
               from2[dimy * (x - 1) + y] = y;
               //printf("[%d][%d]([%d][%d]) expands [%d][%d]([%d][%d])\n", x, y, xg, yg, x - 1, y, xg - 1, yg);
            }
         }

      }
   }

   __syncthreads();

   //return;
   done = 0;

   //Sink to source route tracing
   int xx, yy;
   if(xg == snkx && yg == snky)
   { 
      if(foundSink == 1)
      {    
         success[edgeIndex] = 1;
         xx = x;
         yy = y;
         int found = 0;
         while(!found)
         {
            if(rLeft + xx == srcx && rTop + yy == srcy)
            {
               found = 1;
               break;
            }
         
            graph[gridy * (rLeft + xx) + (rTop + yy)] = wire;

            //Assess top neighbour
            if(yy > 0)
            {
               if(costs[dimy * xx  + (yy - 1)] == costs[dimy * xx + yy] - 1)
               {
                  yy = yy - 1;
                  continue;
               }
            }

            //Assess bottom neighbour
            if(y < dimy)
            {
               if(costs[dimy * xx  + (yy  + 1)] == costs[dimy * xx + yy] - 1)
               {
                  yy = yy + 1;
                  continue;
               }
            }

            //Assess left neighbour
            if(x < dimx)
            {
               if(costs[dimy * (xx + 1) + yy] == costs[dimy * xx + yy] - 1)
               {
                  xx = xx + 1;
                  continue;
               }
            }

            //Assess right neighbour
            if(x > 0)
            {
               if(costs[dimy * (xx - 1) + yy] == costs[dimy * xx + yy] - 1)
               {
                  xx = xx - 1;
                  continue;
               }
            }
         }
      }
   }
   __syncthreads();
   //printf("All here?\n");
   //TODO: Needs to return if the oruting was successful or not
}

void schedule(
   Point **points, 
   Wire *W, 
   vector<vector<pair<int,int>>>edges,
   vector<vector<int>> dependencyList,
   vector<int> routeList,
   vector<BoundingBox> BB,
   int gridx,
   int gridy,
   int numEdges,
   int numWires)
{
   int *graph;

   //TODO: Work out texture memory stuff if there's time
   gpuErrchk(hipMallocManaged(&graph, gridx * gridy * sizeof(int)));
   
   gridToGraph(points, graph, gridx, gridy);

   #ifdef DISPLAY
   drawGrid(gridx, gridy, graph, W);
   #endif

   int srcx, srcy, snkx, snky;
   int rTop; int rBottom; int rLeft; int rRight; //Region boundaries
   int s;

   

   int numSuccessful = 0;
   int attempts = 0;

   //Attempt to route all nets
   while(!routeList.empty())
   {
      s = 0;
      hipStream_t *streams = (hipStream_t *)malloc(numEdges * sizeof(hipStream_t)); 

      int *success;
      gpuErrchk(hipMallocManaged(&success, numEdges * sizeof(int)));

      int edgeIndex = 0;

      for(unsigned int i = 0; i < routeList.size(); i++)
      {
         int ind = routeList[i];
         for(unsigned int j = 0; j < edges[ind].size(); j++)
         {
            
            //Get source and sinx coordinates for this edge
            srcx = W[ind].pins[edges[ind][j].first][0];
            snkx = W[ind].pins[edges[ind][j].second][0];
            srcy = W[ind].pins[edges[ind][j].first][1];
            snky = W[ind].pins[edges[ind][j].second][1];

            //Use the source and sink to create a search region
            rRight = max(srcx, snkx) + 2;
            rRight = rRight >= gridx ? (gridx - 1) : rRight; 

            rLeft = min(srcx, snkx) - 2;
            rLeft = rLeft < 0 ? 0 : rLeft;

            rTop = min(srcy, snky) - 2;
            rTop = rTop < 0 ? 0 : rTop;

            rBottom = max(srcy, snky) + 2;
            rBottom = rBottom >= gridy ? (gridy - 1) : rBottom;

            //Calculate block dimensions for the kernel base don region size
            int dimx = rRight - rLeft;
            int dimy = rBottom - rTop;

            gpuErrchk(hipStreamCreate(&(streams[s])));
            
            dim3 dimBlock(dimx, dimy);
            printf("rTop: %d, rBottom: %d, rLeft: %d, rRight: %d\n", rTop, rBottom, rLeft, rRight);
            printf("dx: %d, dy: %d\n", dimx, dimy);
            printf("src: (%d, %d)  snk: (%d, %d)\n", srcx, srcy, snkx, snky);

            leeMoore<<<1, dimBlock, 0, streams[s++]>>>(srcx, srcy, snkx, snky, 
                                                      rTop, rBottom, rLeft, rRight, 
                                                      gridx, gridy, ind, 
                                                      edgeIndex, success, graph);

            gpuErrchk(hipPeekAtLastError());
            edgeIndex++;
         }
      }

      gpuErrchk(hipDeviceSynchronize());  

      for(int i = 0; i < s; i++)
      {
         gpuErrchk(hipStreamDestroy((streams[i])));
      }

      free(streams);

      #ifdef DISPLAY
      drawGrid(gridx, gridy, graph, W); 
      #endif

      vector<int> successful(numWires);
      vector<int> unsucessful;
      vector<int> done(numWires);

      edgeIndex = 0;
      for(unsigned int i = 0; i < routeList.size(); i++)
      {
         ind = routeList[i];
         done[routeList[i]] = 1;
         successful[i] = 1;
         for(unsigned int j = 0; j < edges[ind].size(); j++)
         {
            if(success[edgeIndex] == 0)
            {
               W[i].found[edges[ind][j].second] = -1; //The sink for this edge hasn't been found
               successful[i] = 0; //If any edges are unsucessful, the wire was unsuccessful
               unsuccessful.push_back(i);
               edgeIndex++;
            }
         }
      }

      free(success);

      vector<vector< std::vector<int>::iterator >> its(dependencyList.size());
      std::vector<int>::iterator it;
      for(unsigned int i = 0; i < dependencyList.size(); i++)
      {
         for(unsigned int j = 0; j < routeList.size(); j++)
         {
            it = std::find(dependencyList[i].begin(), dependencyList[i].end(), routeList[j]);
            if(it != dependencyList[i].end())
            {
               dependencyList[i][*it] = 4096;
            }
         }
      }

      
      for(unsigned int i = 0; i < dependencyList.size(); i++)
      {
         if(dependencyList[i].empty())
         {
            continue;
         }
         dependencyList[i].erase(std::remove_if(dependencyList[i].begin(), dependencyList[i].end(), toDelete), dependencyList[i].end());
      }
      
      routeList.clear();
      numEdges = 0;

      int count = 0;
      for(unsigned int i = 0; i < dependencyList.size(); i++)
      {
         if(dependencyList[i].empty() && done[i] != 1)
         {
               routeList.push_back(i);
               numEdges += edges[i].size();
               count ++;
         }
      }

      printf("concurrent: %d\n", count);
   }

   if(numSuccessful != numWires)
   {
      printf("Not all routed!");
   }
  
}

void gridToGraph(Point **points, int *graph, int gridx, int gridy)
{
   for(int i = 0; i < gridx; i++)
   {
      for(int j = 0; j < gridy; j++)
      {
         graph[gridy * i + j] = points[i][j].obstructedBy;
      }
   }
}

void graphToGrid(Point **points, int *graph, int gridx, int gridy)
{
   for(int i = 0; i < gridx; i++)
   {
      for(int j = 0; j < gridy; j++)
      {
         points[i][j].obstructedBy = graph[gridy * i + j];
      }
   }
}