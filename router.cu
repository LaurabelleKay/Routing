#include <stdlib.h>

#include "router.h"
#include "hip/hip_runtime.h"
#include ""

#include "common.h"
#include <set>
#include <queue>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}